
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    printf("Hello World From GPU!\n");
}

int main(void) {
    printf("Hello World From CPU!\n");
    helloFromGPU<<<1,10>>>();
    hipDeviceReset();
    return 0;
}